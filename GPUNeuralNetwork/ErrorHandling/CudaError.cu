#include "CudaError.cuh"


/**
 * Check if CURAND API call was succesful.
 * If not successful, print where in the code 
 * the error occured and the error status.
*/
void gpu::CudaError::checkCurandError(hiprandStatus_t status, std::string error_message){

    if (status != HIPRAND_STATUS_SUCCESS) {
        std::cout << error_message << std::endl;
        switch (status) {
            case HIPRAND_STATUS_SUCCESS:
            std::cout << "HIPRAND_STATUS_SUCCESS" << std::endl;

            case HIPRAND_STATUS_VERSION_MISMATCH:
            std::cout << "HIPRAND_STATUS_VERSION_MISMATCH" << std::endl;

            case HIPRAND_STATUS_NOT_INITIALIZED:
            std::cout << "HIPRAND_STATUS_NOT_INITIALIZED" << std::endl;

            case HIPRAND_STATUS_ALLOCATION_FAILED:
            std::cout << "HIPRAND_STATUS_ALLOCATION_FAILED" << std::endl;

            case HIPRAND_STATUS_TYPE_ERROR:
            std::cout << "HIPRAND_STATUS_TYPE_ERROR" << std::endl;

            case HIPRAND_STATUS_OUT_OF_RANGE:
            std::cout << "HIPRAND_STATUS_OUT_OF_RANGE" << std::endl;

            case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            std::cout << "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE" << std::endl;

            case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            std::cout << "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED" << std::endl;

            case HIPRAND_STATUS_LAUNCH_FAILURE:
            std::cout << "HIPRAND_STATUS_LAUNCH_FAILURE" << std::endl;

            case HIPRAND_STATUS_PREEXISTING_FAILURE:
            std::cout << "HIPRAND_STATUS_PREEXISTING_FAILURE" << std::endl;

            case HIPRAND_STATUS_INITIALIZATION_FAILED:
            std::cout << "HIPRAND_STATUS_INITIALIZATION_FAILED" << std::endl;

            case HIPRAND_STATUS_ARCH_MISMATCH:
            std::cout << "HIPRAND_STATUS_ARCH_MISMATCH" << std::endl;

            case HIPRAND_STATUS_INTERNAL_ERROR:
            std::cout << "HIPRAND_STATUS_INTERNAL_ERROR" << std::endl;

            default:
                std:: cout << "Unknown CURAND error." << std::endl;
        }
        // Terminate process 
        exit(1);
    }
}