#include "Scalar.cuh"

gpu::Scalar::Scalar(float init_val){
    this->allocateMemHost(init_val);
    this->allocateMemDevice();
    // copy init_val to device
    this->copyHostToDevice();
}

void gpu::Scalar::allocateMemHost(float init_val){
    this->h_scalar = std::make_shared<float>(init_val);
}
void gpu::Scalar::allocateMemDevice(){
    this->d_scalar = std::shared_ptr<float>(nullptr,  [&](float* ptr){ hipFree(ptr);});
    hipMalloc((void**) &this->d_scalar, sizeof(float));
}
void gpu::Scalar::copyHostToDevice(){
    hipMemcpy(this->d_scalar.get(), this->h_scalar.get(), sizeof(float), hipMemcpyHostToDevice);
}
void gpu::Scalar::copyDeviceToHost(){
    hipMemcpy(this->h_scalar.get(), this->d_scalar.get(), sizeof(float), hipMemcpyDeviceToHost);
}

gpu::Scalar& gpu::Scalar::operator=(const gpu::Scalar& rhs){
    // Check if object is being assigned to itself.
    if(this == &rhs){
        return *this;
    }

    this->h_scalar = rhs.h_scalar;
    this->d_scalar = rhs.d_scalar;

    return *this;
}