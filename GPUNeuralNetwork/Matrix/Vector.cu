#include "hip/hip_runtime.h"
#include "Vector.cuh"
#include "Matrix.cuh"
#include <hiprand.h>
#include <iostream>

/**
 * Constructor for Vector object with size of vector and
 * initial values for each element are specified.
 */
gpu::Vector::Vector(int size):
                    m_size(size)
{
    this->allocateMemHost();
    this->allocateMemDevice();
}

/**
 * Constructor for Vector object with initializer list.
 */
gpu::Vector::Vector(std::vector<float> rhs):
                    m_size(rhs.size())
{
    this->allocateMemHost();
    this->allocateMemDevice();

    for(int j= 0; j < this->m_size; j++){
        this->h_vec.get()[j] = rhs[j];
    }

    this->copyHostToDevice();
}

/**
 * TODO
*/
gpu::Vector::Vector(const gpu::Vector& other):
                m_size(other.getSize()),
                h_vec(other.h_vec),
                d_vec(other.d_vec)
{}

/**
 * TODO
*/
void gpu::Vector::allocateMemHost(){
    this->h_vec = std::shared_ptr<float>(new float[this->m_size]{0},
                                        [&](float* ptr){ delete[] ptr; });
}

/**
 * TODO
*/
void gpu::Vector::allocateMemDevice(){
    this->d_vec = std::shared_ptr<float>(nullptr,  [&](float* ptr){ hipFree(ptr);});
    hipMalloc((void**) &this->d_vec, this->m_size*sizeof(float));
}

/**
 * TODO
 */
void gpu::Vector::copyHostToDevice(){
    hipMemcpy(this->d_vec.get(), this->h_vec.get(), this->m_size*sizeof(float), hipMemcpyHostToDevice);
}

/**
 * TODO
 */
void gpu::Vector::copyDeviceToHost(){
    hipMemcpy(this->h_vec.get(), this->d_vec.get(), this->m_size*sizeof(float), hipMemcpyDeviceToHost);
}

/**
 * Compute the dot product between two vectors.
 * 
 * TODO
 * 
 */
__global__ void kDot(float* z, float* W, float* a, int W_size) {

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    float temp = 0.0f;

    if(idx < W_size){
        temp = W[idx]*a[idx]; 
    }

    atomicAdd(z, temp);
}

/**
 * This methode produces a matrix by computing the tensor between two vectors.
 * 
 * TODO
 * 
 */
__global__ void kTensor(float* dLdW, float* a, float* delta, 
                        int dLdW_num_rows, int dLdW_num_cols){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;

    if(idx < dLdW_num_cols && idy < dLdW_num_rows){
        dLdW[idy*dLdW_num_cols + idx] = a[idx]*delta[idy];
    }

}

/**
 * Compute the vector multiplication between a vector and a scalar value.
 * 
 * TODO
 * 
 */
__global__ void kVecScalarMult(float* dLdW, float* a, float delta, int dLdW_size){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < dLdW_size){
        dLdW[idx] = a[idx]*delta;
    }
}

/**
 * 
 * This methode mutiplies a vector by a scalar.
 * The resulting vector is then subtracted from a another vector.
 * 
 */
__global__ void kVecScalarMultSub(float* W, float* dLdW, int W_size){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < W_size){
        W[idx] -= dLdW[idx];
    }
}

__global__ void kVecVecElementwiseMult(float* delta, float* f_prime,  int delta_size){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < delta_size){
        delta[idx] *= f_prime[idx];
    }
}

/**
 * Initialize the elements of the mvector to random values that come
 * from a Gaussian Distribtuion centered at 0 with standard deviations of 
 * @f$\sqrt{ \farc{1}{n_{I}}} $ where @f$n_{I}$ is the size of layer @f$I$.
 * 
 */
void gpu::Vector::vectorInitializationDevice()
{

    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 
                1234ULL);

    float mean = 0.0;
    float stddev = 1/sqrtf(1.0/(float)this->m_size);

    hiprandGenerateNormal(gen, this->d_vec.get(), this->m_size, mean, stddev);

    hiprandDestroyGenerator(gen);

}

/**
 * TODO
*/
gpu::Scalar gpu::Vector::dot(const gpu::Vector& rhs) const{
    gpu::Scalar res(0.0f);

    int threads = 32;
    int blocks = (this->getSize() + threads - 1)/threads;

    kDot<<<blocks, threads>>>(res.d_scalar.get(), this->d_vec.get(), 
                                rhs.d_vec.get(), this->getSize());
    hipDeviceSynchronize();

    return res;

}

/**
 * TODO
*/
gpu::Matrix gpu::Vector::tensor(const Vector& rhs) const{

    int num_rows = rhs.getSize();
    int num_cols = this->m_size;
    gpu::Matrix res(num_rows, num_cols);

    int t = 32;
    int bx = (num_cols + t - 1)/t;
    int by = (num_rows + t - 1)/t;

    dim3 threads(t,t);
    dim3 blocks(bx, by);

    kTensor<<<blocks, threads>>>(res.d_mat.get(), this->d_vec.get(), rhs.d_vec.get(), 
                                  num_rows, num_cols);
    hipDeviceSynchronize();

    return res;
}

/**
 * TODO
*/
void gpu::Vector::deepCopy(gpu::Vector& rhs){
    this->m_size = rhs.getSize();

    rhs.copyDeviceToHost();

    for(int j= 0 ; j < this->m_size; j++){
        this->h_vec.get()[j] = rhs[j];
    }

    this->copyHostToDevice();
}

void gpu::Vector::printVec(){

    this->copyDeviceToHost();

    for (int i=0; i< this->m_size; ++i) {
        std::cout << this->h_vec.get()[i] << std::endl;
    }
}

/**
 * TODO
*/
gpu::Vector& gpu::Vector::operator=(const Vector& rhs){
    // Check if object is being assigned to itself.
    if(this == &rhs){
        return *this;
    }

    this->m_size = rhs.getSize();

    this->h_vec = rhs.h_vec;
    this->d_vec = rhs.d_vec;

    return *this;

}

/**
 * TODO
*/
void gpu::Vector::operator=(const std::vector<float>& rhs){
    this->m_size = rhs.size();


    for(int j= 0 ; j < this->m_size; j++){
        this->h_vec.get()[j] = rhs[j];
    }

    this->copyHostToDevice();
}

/**
 * Overload equality operator.
 * 
 * Two vectora are equal if and only if
 * they have the same size and their
 * corresonding elements are equal.
 * 
 * return true if two vectors are equal,
 *        false otherwise
 */
bool gpu::Vector::operator==(Vector& rhs){
    bool areEqual = true;

    // Variables to store the element of vectors to be compared
    float this_val = 0.0;
    float rhs_val = 0.0;

    // Fixed error for comparison between two given values
    constexpr double epsilon = 0.01; 

    rhs.copyDeviceToHost();
    this->copyDeviceToHost();

    //Check if the sizes of the two vectors are equal
    if( this->m_size != rhs.getSize()){
            areEqual = false;
    }else{
        // Check if corresponding elements of the two vectors are equal
            for(int i = 0; i < this->m_size; i++){
                this_val = this->h_vec.get()[i];
                rhs_val = rhs[i];
                if(!(std::abs(this_val - rhs_val) < epsilon)){
                    areEqual = false;
                }
            }

    }

    return areEqual;
}

/**
 * Overload operator[] for read-only operation on elements of this Vector.
 */
const float gpu::Vector::operator[](const int &input) const{
    return h_vec.get()[input];
}

/**
 * Overload operator[] for write operation on elements of this Vector.
 */
float& gpu::Vector::operator[](const int &input) {
    return h_vec.get()[input];
}

/**
 * TODO
*/
gpu::Vector gpu::Vector::operator*(const float& rhs) const{

    gpu::Vector res(this->m_size);

    int threads = 32;
    int blocks = (this->getSize() + threads -1)/threads;

    kVecScalarMult<<<blocks, threads>>>(res.d_vec.get(), this->d_vec.get(), 
                                        rhs, this->getSize());
    hipDeviceSynchronize();

    return res;

}

gpu::Vector& gpu::Vector::operator*=( const gpu::Vector& rhs){

    int threads = 32;
    int blocks = (this->getSize() + threads - 1)/threads;

    kVecVecElementwiseMult<<<blocks, threads>>>(this->d_vec.get(), rhs.d_vec.get(), this->getSize());
    hipDeviceSynchronize();

    return *this;

}

/**
 * TODO
*/
gpu::Vector& gpu::Vector::operator-=(const Vector& rhs){

    int threads = 32;
    int blocks = (this->getSize() + threads -1)/threads;

    kVecScalarMultSub<<<blocks, threads>>>(this->d_vec.get(), rhs.d_vec.get(), this->getSize());
    hipDeviceSynchronize();

    return *this;

}

/**
 * Return size of this vector.
 */
int gpu::Vector::getSize() const{
    return this->m_size;
}