#include "hip/hip_runtime.h"
#include "HiddenLayer.cuh"

/*=======================*/
// Constructor
/**
 * Constructor for a given hidden layer J in the neural network.
 * 
 * @param layerI_size The number of neurons in the previous layer I.
 * @param layerJ_size The number of neurons in layer J.
 * 
 */
gpu::HiddenLayer::HiddenLayer(int layerI_size, int layerJ_size):
                                m_z(layerJ_size),
                                m_a(layerJ_size),
                                m_delta(layerJ_size),
                                m_W(layerJ_size, layerI_size),
                                m_dLdW(layerJ_size, layerI_size)              
{}

/*=======================*/
// Matrix operations

/**
 * Compute activation of neuron j of layer J using the derivative of ReLu
 * activation function.
 * @f$\begin{math}
        f'(z_j)=\left\{
            \begin{array}{ll}
                0, & \mbox{if $x<0$}.\\
                1, & \mbox{if $x>0$}.
            \end{array}
        \right.
    \end{math}$
 * The derivative is undefined at z_j = 0 but it can be set to zero
 * in order to produce sparse vector.
 * 
 * @param z A value that contains the output of a given neuron i in layer I
 * 
 * @return @f$f'(z_j)$ where @f$z_j$ 
 *         is the output of neuron j of layer J 
 *         and f' is the derivative of the relu activation function.
 * 
 * TODO
 * 
 */
__device__ float reluPrime(int index, float* z){

    float f_prime = (float)(z[index] >= 0);

    return f_prime;

}

/**
 * This methode multiples a matrix with another vector.
 * 
 * TODO
 * 
 */
__global__ void kMatrixVectorMult(float* z, float* W, float* a, int W_num_cols){
    float temp = 0;

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < W_num_cols)
        for(int i=0; i < W_num_cols; i++){
            temp += W[idx*W_num_cols + i]*a[i];
        }
    z[idx] = temp;

}

/**
 * 
 * This methode performs the following operations. 
 * A matrix is first transposed, and then multipled against 
 * a vector. The resulting vector is then multipled by another vector. 
 * 
 * TODO
 * 
 */
__global__ void kMatrixTransposeVectorMult(float* delta, float* W, float* delta_,
                                            float* z, int W_num_rows, int W_num_cols){
    
    float temp;

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < W_num_cols){
        for(int k=0; k < W_num_rows; k++){
            temp += W[idx*W_num_rows + k]*delta_[k];
        }
        delta[idx] = temp*reluPrime(idx, z);
    }

}

/**
 * 
 * This methode performs the following operations.
 * A matrix is multiplied against a vector. 
 * The result is then multiped against another vector.
 * 
 * TODO
 * 
 */
__global__ void kMatrixVectorMult(float* delta, float*W, float delta_, 
                                    float* z, int delta_size){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < delta_size){
        delta[idx] = W[idx]*delta_;
        delta[idx] *= reluPrime(idx, z);
    }
}

/**
 * This methode produces a matrix by computing the tensor between two vectors.
 * 
 * TODO
 * 
 */
__global__ void kTensor(float* dLdW, float* a, float* delta, 
                        int dLdW_num_rows, int dLdW_num_cols){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;

    if(idx < dLdW_num_cols && idy < dLdW_num_rows){
        dLdW[idy*dLdW_num_cols + idx] = a[idx]*delta[idy];
    }

}

/**
 * This methode mutiples a matrix with a scalar.
 * The result is then subtracted from the another matrix.
 * 
 * TODO
 * 
 */
__global__ void kMatrixScalarMultSub(float* W, float* dLdW, float alpha,
                                      int W_num_rows, int W_num_cols){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;

    if(idx < W_num_cols && idy < W_num_rows){
        W[idy*W_num_cols + idx] -= dLdW[idy*W_num_cols+idx]*alpha;
    }

}

__global__ void kReluActivation(float* a, float* z, int a_size){

    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx < a_size){
        a[idx] = fmaxf(0.0, z[idx]);
    }
}

/*=======================*/
// Methodes for forward propegation

/**
 * Initialize the wieghts of this hidden layer.
 */
void gpu::HiddenLayer::weightInitialization(){
    this->m_W.matrixInitializationDevice();
}

/**
 * Compute the output of each neuron j in layer J. 
 * The output for each neuron can be computed as follows 
 * @f$z_j = \sum_{i}^I w_{ji} a_i$ where @f$a_i$ is the output of neuron i
 * from the pervious layer I.
 * 
 * @param a The vector that contains the activations of each neuron in layer I
 * 
 */
void gpu::HiddenLayer::computeOutput(const gpu::Vector &a)
{

    int threads =32;
    int blocks = (this->m_z.getSize() + threads -1)/threads;

    kMatrixVectorMult<<<blocks, threads>>>(this->m_z.d_vec.get(), this->m_W.d_mat.get(), 
                                            a.d_vec.get(), this->m_W.get_num_cols());
    hipDeviceSynchronize();

}

/**
 * Compute the activation of each neuron j in layer J using the ReLu activation function. 
 * The activation for each neuron can be computed as follows 
 * @f$z_j = max(0, z_j)$ where @f$z_j$ is the output of neuron j in layer J.
 * 
 */
void gpu::HiddenLayer::reluActivation()
{

    int threads = 32;
    int blocks = (this->m_a.getSize() + threads -1)/ threads;
    

    kReluActivation<<<blocks, threads>>>(this->m_a.d_vec.get(), this->m_z.d_vec.get(), this->m_a.getSize());
    hipDeviceSynchronize();
}

/**
 * Perform forward propegation on this hidden layer J.
 * 
 * @param a A vector contain the activations of each neuron
 *          in the previous layer.
 * 
 * @return A vector containing the activation of the neurons in 
 *         this hidden layer J.
 * 
 */
gpu::Vector gpu::HiddenLayer::forwardPropegation(const gpu::Vector& a){
    this->computeOutput(a);
    this->reluActivation();

    return this->m_a;
}

/*=======================*/
// Methodes for backward propegation



/**
 * For layers J < K, compute the error term associated with each neuron j of layer J.
 * @f$\delta_j = f'(z_j)\sum_{k=0}^{n_K} w_{kj} \delta_k$ where
 * @f$f'$ is the derivative of the ReLu activation function,
 * @f$z_j$ is the output of neuron j of layer J, @f$n_K$
 * is the number of neurons in layer K, @f$w_{ji}$ is the
 * weight from neuron j of layer J to neuron k of layer K,
 * and @f$\delta_k$ is the error term of neuron k of layer K.
 * 
 * 
 * @param W A vector containing the weigths between layer J and K
 * @param delta The error terms of each neuron k of layer K.
 * 
 */
void gpu::HiddenLayer::computeDelta(const gpu::Vector& W, const float& delta){

    int threads = 32;
    int blocks = (this->m_delta.getSize() + threads - 1)/threads;

    kMatrixVectorMult<<<blocks, threads>>>(this->m_delta.d_vec.get(), W.d_vec.get(), delta,
                                            this->m_z.d_vec.get(), this->m_delta.getSize());
    hipDeviceSynchronize();

}

/**
 * For layers J < K, compute the error term associated with each neuron j of layer J.
 * @f$\delta_j = f'(z_j)\sum_{k=0}^{n_K} w_{kj} \delta_k$ where
 * @f$f'$ is the derivative of the ReLu activation function,
 * @f$z_j$ is the output of neuron j of layer J, @f$n_K$
 * is the number of neurons in layer K, @f$w_{ji}$ is the
 * weight from neuron j of layer J to neuron k of layer K,
 * and @f$\delta_k$ is the error term of neuron k of layer K.
 * 
 * @param W A matrix containing the weigths between layer J and K
 * @param delta_ A vector cotaining the error terms of each neuron k of layer K
 * 
 */
void gpu::HiddenLayer::computeDelta(const gpu::Matrix& W, 
                                    const gpu::Vector& delta){

    int threads = 32;
    int blocks = (this->m_delta.getSize() + threads -1)/threads;
    
    kMatrixTransposeVectorMult<<<blocks, threads>>>(this->m_delta.d_vec.get(), W.d_mat.get(), 
                                                    delta.d_vec.get(), this->m_z.d_vec.get(), 
                                                    W.get_num_rows(), W.get_num_cols());
    hipDeviceSynchronize();

}

/**
 * 
 * Compute the gradient for each weight for a 
 * given layer except the last layer of the neural network.
 * For layers I < J, the gradient for any given weight can be computed as follows.
 * @f$\frac{dL}{dw_{ji}} = a_i * \delta_{j}$ where @f$w_{ji}$ is the weight from
 * neuron i of layer I to neuron j of layer J, @f$a_i$ is the activation of neuron
 * i of layer I, and @f$\delta_{j}$ is the error term of neuron j of layer J.
 * 
 * @param detla A vector containing the error terms for each neuron of layer J
 * @param a     A vector cotaining the activation of each neuron of layer I
 * 
 * 
 */
void gpu::HiddenLayer::computeGradient(const gpu::Vector& a){

    int t = 32;
    int bx = (this->m_dLdW.get_num_cols() + t - 1)/t;
    int by = (this->m_dLdW.get_num_rows() + t - 1)/t;

    dim3 threads(t,t);
    dim3 blocks(bx, by);

    kTensor<<<blocks, threads>>>(this->m_dLdW.d_mat.get(), a.d_vec.get(), this->m_delta.d_vec.get(), 
                                  this->m_dLdW.get_num_rows(), this->m_dLdW.get_num_cols());
    hipDeviceSynchronize();
}

/**
 * Perform Back propegation.
 * 
 * This methode is called to perform back propegation when this hidden layer
 * is the second hidden layer in the neural network.
 * 
 * @return A vector containing the error term for each neuron of this hidden layer.
 */
gpu::Vector gpu::HiddenLayer::backPropegation(const gpu::Vector& W, const float& delta, const gpu::Vector& a){
    this->computeDelta(W, delta);
    this->computeGradient(a);

    return this->m_delta;
}

/**
 * Perform back propegation.
 * 
 * This methode is called to perform back propegation when this 
 * hidden layer is not the last hidden layer in the neural network.
 * 
 * @return A vector containing the error terms for all neurons 
 *         of this hidden layer.
 * 
 */
gpu::Vector gpu::HiddenLayer::backPropegation(const gpu::Matrix& W, const gpu::Vector& delta, const gpu::Vector& a){
    this->computeDelta(W, delta);
    this->computeGradient(a);

    return this->m_delta;
}


/*=======================*/
// Methodes for updating the weights

/**
 * 
 * 
 * Perform gradient descent. For any given weight between layers I < J
 * where Iis the previous layer and J is the output layer,
 * the weight can be updated using the following.
 * @f$ w_{ji} = w_{ji} - \alpha \frac{dL}{dw_{ji}}$
 * 
 * @param alpha The step size of gradient descent
 * 
 */
void gpu::HiddenLayer::gradientDecent(const float& alpha){

    int t = 32;
    int bx = (this->m_dLdW.get_num_cols() + t - 1)/t;
    int by = (this->m_dLdW.get_num_rows() + t - 1)/t;

    dim3 threads(t,t);
    dim3 blocks(bx, by);

    kMatrixScalarMultSub<<<blocks, threads>>>(this->m_W.d_mat.get(), this->m_dLdW.d_mat.get(), alpha,
                                              this->m_W.get_num_rows(), this->m_W.get_num_cols());
    hipDeviceSynchronize();

}

/**
 * 
 * Update weights using gradient descent.
 * 
 */
void gpu::HiddenLayer::updateWeigths(const float& alpha){
    this->gradientDecent(alpha);

}

/*=======================*/

// Getter methods
            
const gpu::Vector& gpu::HiddenLayer::a() const{
    return this->m_a;
}


const gpu::Matrix& gpu::HiddenLayer::W() const{
    return this->m_W;
}

const gpu::Matrix& gpu::HiddenLayer::dLdW() const{
    return this->m_dLdW;
}

// Setter methods

void gpu::HiddenLayer::W(const gpu::Matrix& W){
    this->m_W = W;
}

void gpu::HiddenLayer::WDeepCopy(gpu::Matrix& W){
    this->m_W.deepCopy(W);
}